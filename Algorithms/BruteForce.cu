#include "hip/hip_runtime.h"
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

#include <chrono>

__device__ void generatePermutation(int n, int threadId, char* permutation) {
    // Generate permutation using thread id and n
    for (int i = 0; i < n; i++) {
        permutation[i] = (threadId + i) % n;
    }
}

__device__ void evaluatePermutation(int n, char* permutation, float** graph, int* evaluation, int threadId) {
    // Evaluate permutation
    *evaluation = threadId;
}

__global__ void generateAndEvaluatePermutations(const int n, const unsigned long long int nPermutations,
                                                float** graph, int* bestEvaluation, char* bestPermutation) {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId < nPermutations) {
        char* permutation = (char*)malloc(n * sizeof(char));
        int* evaluation = (int*)malloc(sizeof(int));

        generatePermutation(n, threadId, permutation);
        evaluatePermutation(n, permutation, graph, evaluation, threadId);

        assert(*evaluation == threadId);
        if (*evaluation > *bestEvaluation) {
            
            atomicMax(bestEvaluation, *evaluation);
            *bestPermutation = *permutation;
        }

        free(permutation);
        free(evaluation);
    }
}

char* BruteForce(const int n, int d, float** graph) {
    //
    const unsigned long long int n_permutations = tgamma(n+d+1);

    char* bestPermutation = (char*)malloc(sizeof(char) * (n + d));
    char* threadPermutation;
    int* threadEvaluation;

    constexpr float initialEvaluation = 0;

    hipMalloc(&threadPermutation, sizeof(char) * (n + d)); // Should hold a route permutation
    hipMalloc(&threadEvaluation, sizeof(int));

    hipMemcpy(threadEvaluation, &initialEvaluation, sizeof(float), hipMemcpyHostToDevice);

    constexpr int blockSize = 256;
    int numBlocks = ceil(n_permutations / (float)blockSize);

    generateAndEvaluatePermutations<<<numBlocks, blockSize>>>(n, n_permutations, graph,
                                                              threadEvaluation, threadPermutation);

    hipMemcpy(bestPermutation, threadPermutation, sizeof(char), hipMemcpyDeviceToHost);

    std::cout << "Best permutation: ";
    for (int i = 0; i < n; i++) {
        std::cout << bestPermutation[i] << " ";
    }
    std::cout << std::endl;

    hipFree(threadPermutation);
    hipFree(threadEvaluation);

    return bestPermutation;
}

int main() {
    int n = 3;
    int d = 1;

    char* route = (char*)malloc(sizeof(char) * (n+d));

    // adjacency matrix
    float init[4][4] = {
        {0, 1, 1, 1},
        {1, 0, 1, 1},
        {1, 1, 0, 1},
        {1, 1, 1, 0}
    };

    float** graph = (float**)malloc(n * sizeof(float*));
    for (int i = 0; i < n; i++) {
        graph[i] = init[i];
    }

    route = BruteForce(n, d, graph);

    for (int i = 0; i < (n + d); i++) {
        std::cout << route[i] << " ";
    }
    std::cout << std::endl;

    free(graph);
    free(route);
}

/*
int main()
{
    int n = 50;

    auto serial_begin = std::chrono::high_resolution_clock::now();
    unsigned long long int serial_result = tgamma(n+1);
    auto serial_end = std::chrono::high_resolution_clock::now();

    std::chrono::high_resolution_clock::time_point parallel_begin = std::chrono::high_resolution_clock::now();
    std::vector<int> n_vector(n-1);
    std::iota(n_vector.begin(), n_vector.end(), 2);
    unsigned long long int parallel_result = thrust::reduce(thrust::host, n_vector.begin(), n_vector.end(),
        1, thrust::multiplies<int>());
    std::chrono::high_resolution_clock::time_point parallel_end = std::chrono::high_resolution_clock::now();

    auto serial_time_taken = serial_end - serial_begin;
    auto parallel_time_taken = parallel_end - parallel_begin;

    std::cout << "Serial result: " << serial_result << std::endl;
    std::cout << "Parallel result: " << parallel_result << std::endl << std::endl;

    std::cout << "Serial time: " << std::chrono::duration<double>(serial_time_taken).count() << std::endl;
    std::cout << "Parallel time: " << std::chrono::duration<double>(parallel_time_taken).count() << std::endl;

    return 0;
}*/